#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.cuh"

// A dummy kernel that prints block and thread indices
__global__ void dummyKernel() {
    printf("Block %d, Thread %d\n", blockIdx.x, threadIdx.x);
}