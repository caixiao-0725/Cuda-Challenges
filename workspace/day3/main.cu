#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

void vectorAddCPU(float* A, float* B, float* C, int N) {
    for (int i = 0; i < N; i++) {
        C[i] = A[i] + B[i];
    }
}

__global__ void vectorAddGPU(float* A, float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 100000000; // 100 million
    float* A = (float*)malloc(N * sizeof(float));
    float* B = (float*)malloc(N * sizeof(float));
    float* C = (float*)malloc(N * sizeof(float));

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    clock_t CPU_start = clock();
    vectorAddCPU(A, B, C, N);
    clock_t CPU_end = clock();

    printf("CPU Vector Add Time: %lf seconds\n", ((double)(CPU_end - CPU_start)) / CLOCKS_PER_SEC);

    free(A);
    free(B);
    free(C);




    size_t size = N * sizeof(float);
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float* d_A, * d_B, * d_C;
    hipError_t err;

    err = hipMalloc(&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for A: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for B: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    clock_t GPU_start = clock();
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddGPU << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        exit(EXIT_FAILURE);
    }
    clock_t GPU_end = clock();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("GPU Vector Add Time: %lf seconds\n", ((double)(GPU_end - GPU_start)) / CLOCKS_PER_SEC);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;

}